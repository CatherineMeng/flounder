#include "hip/hip_runtime.h"
#include "ints.cuh"
#include "mycuda.cuh"
#include <assert.h>
#include <limits.h>
#if 0
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#endif

struct ints ints_new(int n)
{
  struct ints is;
  is.n = n;
  CUDACALL(hipMalloc(&is.i, sizeof(int) * n));
  return is;
}

void ints_free(struct ints is)
{
  CUDACALL(hipFree(is.i));
}

struct ints ints_exscan(struct ints is)
{
  struct ints o = ints_new(is.n + 1);
#if 0
  thrust::device_ptr<int> inp(is.i);
  thrust::device_ptr<int> outp(o.i);
  thrust::exclusive_scan(inp, inp + is.n, outp);
  /* fixup the last element quirk */
  int sum = thrust::reduce(inp, inp + is.n);
  CUDACALL(hipMemcpy(o.i + is.n, &sum, sizeof(int), hipMemcpyHostToDevice));
#else
  int* hi = (int*) malloc(sizeof(int) * (is.n + 1));
  int* ho = (int*) malloc(sizeof(int) * (is.n + 1));
  CUDACALL(hipMemcpy(hi, is.i, sizeof(int) * is.n, hipMemcpyDeviceToHost));
  int sum = 0;
  ho[0] = 0;
  for (int i = 0; i < is.n; ++i) {
    sum += hi[i];
    ho[i + 1] = sum;
  }
  CUDACALL(hipMemcpy(o.i, ho, sizeof(int) * (is.n + 1), hipMemcpyHostToDevice));
  free(hi);
  free(ho);
#endif
  return o;
}

int ints_max(struct ints is)
{
#if 0
  thrust::device_ptr<int> p(is.i);
  return thrust::reduce(p, p + is.n, INT_MIN, thrust::maximum<int>());
#else
  int* hi = (int*) malloc(sizeof(int) * is.n);
  CUDACALL(hipMemcpy(hi, is.i, sizeof(int) * is.n, hipMemcpyDeviceToHost));
  int max = hi[0];
  for (int i = 1; i < is.n; ++i)
    if (hi[i] > max)
      max = hi[i];
  free(hi);
  return max;
#endif
}

void ints_zero(struct ints is)
{
  CUDACALL(hipMemset(is.i, 0, sizeof(int) * is.n));
}

void ints_copy(struct ints into, struct ints from, int n)
{
  CUDACALL(hipMemcpy(into.i, from.i, sizeof(int) * n, hipMemcpyDeviceToDevice));
}

void ints_from_host(struct ints is, int const host_dat[])
{
  CUDACALL(hipMemcpy(is.i, host_dat, sizeof(int) * is.n, hipMemcpyHostToDevice));
}
