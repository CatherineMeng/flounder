#include "hip/hip_runtime.h"
#include "ints.cuh"
#include "mycuda.cuh"
#include <assert.h>
#include <limits.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

struct ints ints_new(int n)
{
  struct ints is;
  is.n = n;
  CUDACALL(hipMalloc(&is.i, sizeof(int) * n));
  return is;
}

void ints_free(struct ints is)
{
  CUDACALL(hipFree(is.i));
}

struct ints ints_exscan(struct ints is)
{
  struct ints o = ints_new(is.n + 1);
  thrust::device_ptr<int> inp(is.i);
  thrust::device_ptr<int> outp(o.i);
  thrust::exclusive_scan(inp, inp + is.n, outp);
  /* fixup the last element quirk */
  int sum = thrust::reduce(inp, inp + is.n);
  CUDACALL(hipMemcpy(o.i + is.n, &sum, sizeof(int), hipMemcpyHostToDevice));
  return o;
}

int ints_max(struct ints is)
{
  thrust::device_ptr<int> p(is.i);
  int max = thrust::reduce(p, p + is.n, INT_MIN, thrust::maximum<int>());
  CUDACALL2(hipDeviceSynchronize());
  return max;
}

void ints_zero(struct ints is)
{
  CUDACALL(hipMemset(is.i, 0, sizeof(int) * is.n));
}

void ints_copy(struct ints into, struct ints from, int n)
{
  CUDACALL(hipMemcpy(into.i, from.i, sizeof(int) * n, hipMemcpyDeviceToDevice));
}

void ints_from_host(struct ints is, int const host_dat[])
{
  CUDACALL(hipMemcpy(is.i, host_dat, sizeof(int) * is.n, hipMemcpyHostToDevice));
}

void ints_to_host(struct ints* is)
{
  int* tmp = (int*) malloc(sizeof(int) * is->n);
  CUDACALL(hipMemcpy(tmp, is->i, sizeof(int) * is->n, hipMemcpyDeviceToHost));
  CUDACALL(hipFree(is->i));
  is->i = tmp;
}

void ints_to_device(struct ints* is)
{
  int* tmp;
  CUDACALL(hipMalloc(&tmp, sizeof(int) * is->n));
  CUDACALL(hipMemcpy(tmp, is->i, sizeof(int) * is->n, hipMemcpyHostToDevice));
  free(is->i);
  is->i = tmp;
}
